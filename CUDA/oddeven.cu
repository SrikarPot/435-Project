#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <caliper/cali.h>
#include <stdlib.h>
#include <time.h>
#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

int THREADS;
int BLOCKS;
int NUM_VALS;

// Define the array size
#define N 8

__device__ void device_array_print(float *arr, int length)
{
  int i;
  for (i = 0; i < length; ++i)
  {
    printf("%1.3f ", arr[i]);
  }
  printf("\n");
}

void array_print(float *arr, int length)
{
  int i;
  for (i = 0; i < length; ++i)
  {
    printf("%1.3f ", arr[i]);
  }
  printf("\n");
}

// CUDA kernel function for odd-even transposition sort
__global__ void oddEvenSortKernel(float *d_a, int n, int phase)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int idx1 = index;
  int idx2 = index + 1;
  printf("%d\n", idx1);
  // Check whether we are in an odd or even phase
  if ((phase % 2 == 0) && (idx2 < n) && (idx1 % 2 == 0))
  { // Even phase
    if (d_a[idx1] > d_a[idx2])
    {
      // Swap elements
      float temp = d_a[idx1];
      d_a[idx1] = d_a[idx2];
      d_a[idx2] = temp;
    }
  }
  else if ((phase % 2 == 1) && (idx2 < n) && (idx1 % 2 == 1))
  { // Odd phase
    if (d_a[idx1] > d_a[idx2])
    {
      // Swap elements
      float temp = d_a[idx1];
      d_a[idx1] = d_a[idx2];
      d_a[idx2] = temp;
    }
  }
}

// Host function to run the odd-even transposition sort
void cudaOddEvenSort(float *h_a, int n)
{
  float *d_a;
  // Allocate memory on the device
  hipMalloc(&d_a, n * sizeof(int));
  // Copy data from host to device
  CALI_MARK_BEGIN("comm");
  CALI_MARK_BEGIN("comm_large");
  hipMemcpy(d_a, h_a, n * sizeof(int), hipMemcpyHostToDevice);
  CALI_MARK_END("comm_large");
  CALI_MARK_END("comm");

  // Setup block and grid dimensions
  dim3 blocks(BLOCKS, 1);   /* Number of blocks   */
  dim3 threads(THREADS, 1); /* Number of threads  */

  // Caliper instrumentation for computation region
  // CALI_MARK_BEGIN("comp");
  // CALI_MARK_BEGIN("comp_large");

  // Launch the kernel multiple times
  CALI_MARK_BEGIN("comp");
  CALI_MARK_BEGIN("comp_large");
  for (int i = 0; i < n; ++i)
  {
    oddEvenSortKernel<<<blocks, threads>>>(d_a, n, i);
    hipDeviceSynchronize();
    hipMemcpy(h_a, d_a, n * sizeof(int), hipMemcpyDeviceToHost);
    array_print(h_a, n);
  }
  CALI_MARK_END("comp_large");
  CALI_MARK_END("comp");

  // CALI_MARK_END("comp_large");
  // CALI_MARK_END("comp");

  // Copy the sorted array back to the host
  hipMemcpy(h_a, d_a, n * sizeof(int), hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(d_a);
}

// Function to initialize data in the array
void data_init(int *h_a, int n)
{
  int init_data[N] = {7, 3, 5, 8, 2, 9, 4, 1};
  for (int i = 0; i < n; i++)
  {
    h_a[i] = init_data[i];
  }
}

// Function to check the correctness of the sort
int correctness_check(float *h_a, int n)
{
  for (int i = 1; i < n; i++)
  {
    if (h_a[i - 1] > h_a[i])
    {
      return 0; // Array is not sorted correctly
    }
  }
  return 1; // Array is sorted correctly
}

float random_float()
{
  return (float)rand() / (float)RAND_MAX;
}

void array_fill(float *arr, int length)
{
  srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i)
  {
    arr[i] = random_float();
  }
}

int main(int argc, char *argv[])
{

  THREADS = atoi(argv[1]);  // Number of threads
  NUM_VALS = atoi(argv[2]); // Number of values in the array
  BLOCKS = NUM_VALS / THREADS;
  float *values = (float *)malloc(NUM_VALS * sizeof(float));

  // CALI_MARK_BEGIN("main");

  printf("Number of threads: %d\n", THREADS);
  printf("Number of values: %d\n", NUM_VALS);
  printf("Number of blocks: %d\n", BLOCKS);

  // Initialize data in the host array
  // CALI_MARK_BEGIN("data_init");
  // data_init(h_a, N);
  array_fill(values, NUM_VALS);
  array_print(values, NUM_VALS);

  cali::ConfigManager mgr;
  mgr.start();

  // CALI_MARK_END("data_init");

  // Caliper annotation for communication region, for example with MPI (not present in the code)
  // CALI_MARK_BEGIN("comm");

  // CALI_MARK_END("comm");

  // Caliper instrumentation for computation region
  CALI_MARK_BEGIN("comp");
  CALI_MARK_BEGIN("comp_large");
  // Perform sorting on the GPU
  cudaOddEvenSort(values, NUM_VALS);
  hipDeviceSynchronize();

  CALI_MARK_END("comp_large");
  CALI_MARK_END("comp");

  // Caliper annotation for checking the correctness of the sorting operation
  CALI_MARK_BEGIN("correctness_check");
  int is_correct = correctness_check(values, NUM_VALS);
  CALI_MARK_END("correctness_check");

  if (is_correct)
  {
    printf("The array is sorted correctly.\n");
  }
  else
  {
    printf("The array is NOT sorted correctly.\n");
  }

  // CALI_MARK_END("main");

  array_print(values, 5);

  // Thicket.tree();
  mgr.stop();
  mgr.flush();

  return 0;
}