#include "hip/hip_runtime.h"
/*
 * Parallel bitonic sort using CUDA.
 * Compile with
 * nvcc bitonic_sort.cu
 * Based on http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
 * License: BSD 3
 */

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>
#include "../helper.h"

int THREADS;
int BLOCKS;
int NUM_VALS;
int kernelCalls = 0;

const char* bitonic_sort_step_region = "bitonic_sort_step";
const char* cudaMemcpy_host_to_device = "cudaMemcpy_host_to_device";
const char* cudaMemcpy_device_to_host = "cudaMemcpy_device_to_host";

// Store results in these variables.
float effective_bandwidth_gb_s;
float bitonic_sort_step_time;
float cudaMemcpy_host_to_device_time;
float cudaMemcpy_device_to_host_time;




__device__ void merge(float* values, float* temp, int l, int m, int r)  {
  int i = l;
  int j = m;
  int k = l;

  while(i < m && j < r) {
    if(values[i] > values[j])
      temp[k++] = values[j++];
    else
      temp[k++] = values[i++];
  }

  // add left over values from first half
  while(i < m) {
    temp[k++] = values[i++];
  }

  //add left over values from second half
  while(j < r) {
    temp[k++] = values[j++];
  }

  // copy over to main array
  for(i = l; i < r; i++) {
    values[i] = temp[i];
  }

}


__global__ void merge_sort(float* values, float* temp, int num_vals, int window) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int l = id*window;
    int r = l+window;

    if(r > num_vals) { // final window might be smaller
      r = num_vals;
    }

    int m = l + (r-l)/2;  

    if(l < num_vals) { // check if thread is neccesary
      merge(values, temp, l, m, r);
    }
} 


/**
 * Inplace merge sort using CUDA.
 */
void merge_sort_caller(float *values)
{
  float *dev_values, *temp;
  int size = NUM_VALS * sizeof(float);

  hipMalloc((void**)&dev_values, size);
  hipMalloc((void**)&temp, size);
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);


  
  dim3 blocks(BLOCKS,1);    /* Number of blocks   */
  dim3 threads(THREADS,1);  /* Number of threads  */

  for(int window = 2; window <= size; window <<=1) {
    merge_sort<<<blocks, threads>>>(dev_values, temp, NUM_VALS, window);
  }

  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  hipFree(dev_values);
  hipFree(temp);
  
}

int main(int argc, char *argv[])
{
  CALI_MARK_BEGIN("main");
  THREADS = atoi(argv[1]);
  NUM_VALS = atoi(argv[2]);
  BLOCKS = NUM_VALS / THREADS;

  printf("Number of values: %d\n", NUM_VALS);
  printf("Number of values: %d\n", NUM_VALS);
  printf("Number of blocks: %d\n", BLOCKS);
  // Create caliper ConfigManager object
  cali::ConfigManager mgr;
  mgr.start();

  clock_t start, stop;
  CALI_MARK_BEGIN("data_init");
  float *values = (float*) malloc( NUM_VALS * sizeof(float));
  array_fill(values, NUM_VALS);
  CALI_MARK_END("data_init");

  start = clock();
  merge_sort_caller(values); /* Inplace */
  stop = clock();

  print_elapsed(start, stop);
  array_print(values, NUM_VALS);
//   double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
//   effective_bandwidth_gb_s = kernelCalls*6*NUM_VALS*sizeof(float)/1e9/elapsed;

//   printf("kernel calls: %d\n", kernelCalls);
//   printf("cudaMemcpy_host_to_device_time: %f\n", cudaMemcpy_host_to_device_time/1000);
//   printf("cudaMemcpy_device_to_host_time: %f\n", cudaMemcpy_device_to_host_time/1000);
//   printf("bitonic_sort_step_time: %f\n", bitonic_sort_step_time/1000);
//   printf("effective_bandwitdth_gb_s: %f\n", effective_bandwidth_gb_s);

//   adiak::init(NULL);
//   adiak::user();
//   adiak::launchdate();
//   adiak::libraries();
//   adiak::cmdline();
//   adiak::clustername();
//   adiak::value("num_threads", THREADS);
//   adiak::value("num_blocks", BLOCKS);
//   adiak::value("num_vals", NUM_VALS);
//   adiak::value("program_name", "cuda_bitonic_sort");
//   adiak::value("datatype_size", sizeof(float));
//   adiak::value("effective_bandwidth (GB/s)", effective_bandwidth_gb_s);
//   adiak::value("bitonic_sort_step_time", bitonic_sort_step_time);
//   adiak::value("cudaMemcpy_host_to_device_time", cudaMemcpy_host_to_device_time);
//   adiak::value("cudaMemcpy_device_to_host_time", cudaMemcpy_device_to_host_time);
  CALI_MARK_END("main");

  // Flush Caliper output before finalizing MPI
  mgr.stop();
  mgr.flush();
}