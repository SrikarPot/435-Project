#include "hip/hip_runtime.h"
/*
 * Parallel bitonic sort using CUDA.
 * Compile with
 * nvcc bitonic_sort.cu
 * Based on http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
 * License: BSD 3
 */

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <iostream>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int THREADS;
int BLOCKS;
int NUM_VALS;

hipEvent_t start_sort, end_sort, start_host_device, end_host_device, start_device_host, end_device_host;

// Store results in these variables.
float effective_bandwidth_gb_s = 0;
float bitonic_sort_step_time = 0;
float cudaMemcpy_host_to_device_time = 0;
float cudaMemcpy_device_to_host_time = 0;

void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.3fs\n", elapsed);
}

float random_float()
{
  return (float)rand()/(float)RAND_MAX;
}

void array_print(float *arr, int length) 
{
  int i;
  for (i = 0; i < length; ++i) {
    printf("%1.3f ",  arr[i]);
  }
  printf("\n");
}

void array_fill(float *arr, int length)
{
  srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i) {
    arr[i] = random_float();
  }
}

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (dev_values[i]>dev_values[ixj]) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (dev_values[i]<dev_values[ixj]) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}

/**
 * Inplace bitonic sort using CUDA.
 */
void bitonic_sort(float *values)
{
  float *dev_values;
  size_t size = NUM_VALS * sizeof(float);

  hipMalloc((void**) &dev_values, size);
  
  //MEM COPY FROM HOST TO DEVICE
  CALI_MARK_BEGIN("comm");
  CALI_MARK_BEGIN("comm_large");
  hipEventRecord(start_host_device);
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);
  hipEventRecord(end_host_device);
  CALI_MARK_END("comm_large");
  CALI_MARK_END("comm");

  hipEventSynchronize(end_host_device);
  hipEventElapsedTime(&cudaMemcpy_host_to_device_time, start_host_device, end_host_device);

  dim3 blocks(BLOCKS,1);    /* Number of blocks   */
  dim3 threads(THREADS,1);  /* Number of threads  */
  
  int j, k;
  int kernel_call = 1;
  CALI_MARK_BEGIN("comp");
  CALI_MARK_BEGIN("comp_large");
  hipEventRecord(start_sort);
  /* Major step */
  for (k = 2; k <= NUM_VALS; k <<= 1) {
    /* Minor step */
    for (j=k>>1; j>0; j=j>>1) {
      bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
      kernel_call++;
    }
  }
  hipEventRecord(end_sort);
  hipDeviceSynchronize();
  CALI_MARK_END("comp_large");
  CALI_MARK_END("comp");
  hipEventElapsedTime(&bitonic_sort_step_time, start_sort, end_sort);

  effective_bandwidth_gb_s = ((kernel_call * 6 * NUM_VALS * sizeof(float)) / 1e9) / (bitonic_sort_step_time / 1000);
  
  
  //MEM COPY FROM DEVICE TO HOST
  CALI_MARK_BEGIN("comm");
  CALI_MARK_BEGIN("comm_large");
  hipEventRecord(start_device_host);
  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  hipEventRecord(end_device_host);
  CALI_MARK_END("comm_large");
  CALI_MARK_END("comm");

  hipEventSynchronize(end_device_host);
  hipEventElapsedTime(&cudaMemcpy_device_to_host_time, start_device_host, end_device_host);
  
  hipFree(dev_values);
}

int main(int argc, char *argv[])
{
  hipEventCreate(&start_sort);
  hipEventCreate(&end_sort);
  hipEventCreate(&start_host_device);
  hipEventCreate(&end_host_device);
  hipEventCreate(&start_device_host);
  hipEventCreate(&end_device_host);
  THREADS = atoi(argv[1]);
  NUM_VALS = atoi(argv[2]);
  BLOCKS = NUM_VALS / THREADS;

  printf("Number of threads: %d\n", THREADS);
  printf("Number of values: %d\n", NUM_VALS);
  printf("Number of blocks: %d\n", BLOCKS);

  // Create caliper ConfigManager object
  cali::ConfigManager mgr;
  mgr.start();

  clock_t start, stop;

  float *values = (float*) malloc( NUM_VALS * sizeof(float));
  array_fill(values, NUM_VALS);

  start = clock();
  bitonic_sort(values); /* Inplace */
  stop = clock();

  print_elapsed(start, stop);

  std::cout << "bitonic sort step time: " << bitonic_sort_step_time << std::endl;
  std::cout << "host to device time: " << cudaMemcpy_host_to_device_time << std::endl;
  std::cout << "device to host time: " << cudaMemcpy_device_to_host_time << std::endl;
  std::cout << "effective bandwidth: " << effective_bandwidth_gb_s << std::endl;


  adiak::init(NULL);
  adiak::user();
  adiak::launchdate();
  adiak::libraries();
  adiak::cmdline();
  adiak::clustername();
  adiak::value("num_threads", THREADS);
  adiak::value("num_blocks", BLOCKS);
  adiak::value("num_vals", NUM_VALS);
  adiak::value("program_name", "cuda_bitonic_sort");
  adiak::value("datatype_size", sizeof(float));
  adiak::value("effective_bandwidth (GB/s)", effective_bandwidth_gb_s);
  adiak::value("bitonic_sort_step_time", bitonic_sort_step_time);
  adiak::value("cudaMemcpy_host_to_device_time", cudaMemcpy_host_to_device_time);
  adiak::value("cudaMemcpy_device_to_host_time", cudaMemcpy_device_to_host_time);

  // Flush Caliper output before finalizing MPI
  mgr.stop();
  mgr.flush();
}