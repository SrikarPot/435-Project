#include "hip/hip_runtime.h"
/*
 * Parallel bitonic sort using CUDA.
 * Compile with
 * nvcc bitonic_sort.cu
 * Based on http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
 * License: BSD 3
 */

#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <hip/hip_runtime.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

int THREADS;
int BLOCKS;
int NUM_VALS;

const char* bitonic_sort_step_region = "bitonic_sort_step";
const char* cudaMemcpy_host_to_device = "cudaMemcpy_host_to_device";
const char* cudaMemcpy_device_to_host = "cudaMemcpy_device_to_host";

/* Define Caliper region names */
const char* comm = "comm";
const char* comm_large = "comm_large";
const char* comp = "comp";
const char* comp_large = "comp_large";

int bitonic_counter = 0;

void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.3fs\n", elapsed);
}

float random_float()
{
  return (float)rand()/(float)RAND_MAX;
}

void array_print(float *arr, int length) 
{
  int i;
  for (i = 0; i < length; ++i) {
    printf("%1.3f ",  arr[i]);
  }
  printf("\n");
}

void array_fill(float *arr, int length)
{
  srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i) {
    arr[i] = random_float();
  }
}

__global__ void enumerationSort(float *array, int *rank, int n, int THREADS) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = k; i < n; i += THREADS){
        
        if (i < n) {
            rank[i] = 0;
            for (int j = 0; j < n; j++) {
                if (array[j] < array[i] || (array[j] == array[i] && j < i)) {
                    rank[i]++;
                }
            }
        }
    }
}

// Helper function to swap two integers
__device__ void swap(int &a, int &b) {
    int temp = a;
    a = b;
    b = temp;
}

// CUDA kernel for sorting the array based on ranks
__global__ void sortArray(float *array, float *sorted_array, int *rank, int n, int THREADS) {
    for (int i = 0; i < n; i += THREADS){
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    // for(int i = k; i < n; i += THREADS){
    //     if (i < n) {
    //         for (int j = 0; j < n; j++) {
    //             if (rank[j] == i) {
    //                 swap(array[j], array[i]);
    //                 swap(rank[j], rank[i]);
    //                 break;
    //             }
    //         }
    //     }
    // }
    for (int i = k; i < n; i += THREADS){
        sorted_array[rank[i]] = array[i];
    }
}


int main(int argc, char *argv[])
{
  

    THREADS = atoi(argv[1]);
    NUM_VALS = atoi(argv[2]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n", BLOCKS);

    // Create caliper ConfigManager object
    cali::ConfigManager mgr;
    mgr.start();


    const int n = NUM_VALS; // Size of the array
    float *h_array = new float[n];
    int *h_rank = new int[n];
    float *sorted_array = new float[n];

    // Initialize the array with random values
    array_fill(h_array, n);
    

    // Print the og array
    std::cout << "Original Array: ";
    for (int i = 0; i < n; i++) {
        std::cout << h_array[i] << " ";
    }
    std::cout << std::endl;

    // Device arrays
    float *d_array, *sorted_array_device;
    int* d_rank;
    hipMalloc((void**)&d_array, sizeof(float) * n);
    hipMalloc((void**)&d_rank, sizeof(int) * n);
    hipMalloc((void**)&sorted_array_device, sizeof(float) * n);

    CALI_MARK_BEGIN("comm");
    CALI_MARK_BEGIN("comm_large");

    // Copy data from host to device
    hipMemcpy(d_array, h_array, sizeof(float) * n, hipMemcpyHostToDevice);

    CALI_MARK_BEGIN("comm_large");
    CALI_MARK_BEGIN("comm");

    CALI_MARK_BEGIN("comp");
    CALI_MARK_BEGIN("comp_large");

    // Launch the enumeration sort kernel
    enumerationSort<<<BLOCKS, THREADS>>>(d_array, d_rank, n, THREADS);
    hipDeviceSynchronize();
    CALI_MARK_END("comp_large");

    CALI_MARK_BEGIN("comp_large");
    // Launch the sorting kernel to rearrange the array
    sortArray<<<BLOCKS, THREADS>>>(d_array, sorted_array_device, d_rank, n, THREADS);
    hipDeviceSynchronize();

    CALI_MARK_END("comp_large");
    CALI_MARK_END("comp");

    CALI_MARK_BEGIN("comm");
    
    // Copy the sorted array and ranks back to the host
    // CALI_MARK_BEGIN("comm_large");
    // hipMemcpy(h_array, d_array, sizeof(float) * n, hipMemcpyDeviceToHost);
    // CALI_MARK_END("comm_large");

    // CALI_MARK_BEGIN("comm_large");
    // hipMemcpy(h_rank, d_rank, sizeof(int) * n, hipMemcpyDeviceToHost);
    // CALI_MARK_END("comm_large");

    CALI_MARK_BEGIN("comm_large");
    hipMemcpy(sorted_array, sorted_array_device, sizeof(float) * n, hipMemcpyDeviceToHost);
    CALI_MARK_END("comm_large");

    CALI_MARK_END("comm");

    // for (int i = 0; i < NUM_VALS; i++){
    //     sorted_array[rank[i]] = h_array[i];
    // }

    // Print the sorted array
    std::cout << "Sorted Array: ";
    for (int i = 0; i < n; i++) {
        std::cout << sorted_array[i] << " ";
    }
    std::cout << std::endl;
 
    // Clean up
    delete[] h_array;
    delete[] h_rank;
    delete[] sorted_array;
    hipFree(d_array);
    hipFree(d_rank);
    hipFree(sorted_array_device);



    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", "EnumerationSort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "CUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", float); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", 4); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", NUM_VALS); // The number of elements in input dataset (1000)
    adiak::value("InputType", "Sorted"); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    // adiak::value("num_procs", ); // The number of processors (MPI ranks)
    adiak::value("num_threads", THREADS); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", BLOCKS); // The number of CUDA blocks 
    adiak::value("group_num", 15); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "Handwritten"); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").



//   print_elapsed(start, stop);


  // Flush Caliper output before finalizing MPI
  mgr.stop();
  mgr.flush();
}